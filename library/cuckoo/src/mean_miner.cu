#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2018 John Tromp
// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html
// xenoncat demonstrated at https://github.com/xenoncat/cuckoo_pow
// how bucket sorting avoids random memory access latency
// This CUDA port of mean_miner.cpp is covered by the FAIR MINING license

#include "cuckoo.h"
#include "siphash.cuh"
#include <sys/time.h> // gettimeofday
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <vector>
#include <bitset>

// The node bits are logically split into 3 groups:
// XBITS 'X' bits (most significant), YBITS 'Y' bits, and ZBITS 'Z' bits (least significant)
// Here we have the default XBITS=YBITS=7, ZBITS=15 summing to EDGEBITS=29
// nodebits   XXXXXXX YYYYYYY ZZZZZZZZZZZZZZZ
// bit%10     8765432 1098765 432109876543210
// bit/10     2222222 2111111 111110000000000

// The matrix solver stores all edges in a matrix of NX * NX buckets,
// where NX=2^XBITS is the number of possible values of the 'X' bits.
// Edge i between nodes ui = siphash24(2*i) and vi = siphash24(2*i+1)
// resides in the bucket at (uiX,viX)
// In each trimming round, either a matrix row or a matrix column (NX buckets)
// is bucket sorted on uY or vY respectively, and then within each bucket
// uZ or vZ values are counted and edges with a count of only one are eliminated,
// while remaining edges are bucket sorted back on vY or uY respectively.
// When sufficiently many edges have been eliminated, a pair of compression
// rounds remap surviving Z values in each X,Y bucket to fit into 16-YBITS bits,
// allowing the remaining rounds to avoid the sorting on Y and directly
// count YZ values in a cache friendly 16KB.

// algorithm/performance parameters
// EDGEBITS/NEDGES/EDGEMASK defined in cuckoo.h

#ifndef XBITS
// 7 seems to give best performance
#define XBITS 7
#endif

#define YBITS XBITS

// size in bytes of a big bucket entry
#ifndef BIGSIZE
#define BIGSIZE 5
#endif

// YZ compression round; must be even
#ifndef COMPRESSROUND
#define COMPRESSROUND 10
#endif

typedef uint8_t u8;
typedef uint16_t u16;

const static u32 NX        = 1 << XBITS;
const static u32 XMASK     = NX - 1;
const static u32 NY        = 1 << YBITS;
const static u32 YMASK     = NY - 1;
const static u32 XYBITS    = XBITS + YBITS;
const static u32 NXY       = 1 << XYBITS;
const static u32 ZBITS     = EDGEBITS - XYBITS;
const static u32 ZBYTES    = (ZBITS + 7) / 8;
const static u32 NZ        = 1 << ZBITS;
const static u32 ZMASK     = NZ - 1;
const static u32 YZBITS    = YBITS + ZBITS;
const static u32 NYZ       = 1 << YZBITS;
const static u32 YZMASK    = NYZ - 1;
const static u32 YZ1BITS   = 16;  // make UYZ1,VYZ1 fit in 32 bits
const static u32 NYZ1      = 1 << YZ1BITS;
const static u32 MAXNZNYZ1 = NYZ1 > NZ ? NYZ1 : NZ;
const static u32 YZ1MASK   = NYZ1 - 1;
const static u32 Z1BITS    = YZ1BITS - YBITS;
const static u32 NZ1       = 1 << Z1BITS;
const static u32 Z1MASK    = NZ1 - 1;
const static u32 YZ2BITS   = 9;  // more compressed edge reduces CUCKOO size
const static u32 NYZ2      = 1 << YZ2BITS;
const static u32 CUCKOO_SIZE = 2 * NX * NYZ2; // 2^17 elem  cuckoo table
const static u32 XYZ2BITS  = XBITS + YZ2BITS;
const static u32 XYZ2MASK  = (1 << XYZ2BITS) - 1;
const static u32 Z2BITS    = YZ2BITS - YBITS;
const static u32 NZ2       = 1 << Z2BITS;
const static u32 Z2MASK    = NZ2 - 1;
const static u32 YZZBITS   = YZBITS + ZBITS;
const static u32 YZZ1BITS  = YZ1BITS + ZBITS;

const static u32 BIGSLOTBITS   = BIGSIZE * 8;
const static u32 NONYZBITS     = BIGSLOTBITS - YZBITS;
const static u32 NNONYZ        = 1 << NONYZBITS;

const static u32 Z2BUCKETSIZE = NYZ2 * sizeof(u32);

// for p close to 0, Pr(X>=k) < e^{-n*p*eps^2} where k=n*p*(1+eps)
// see https://en.wikipedia.org/wiki/Binomial_distribution#Tail_bounds
// eps should be at least 1/sqrt(n*p/64)
// to give negligible bad odds of e^-64.

// 1/32 reduces odds of overflowing z bucket on 2^30 nodes to 2^14*e^-32
// (less than 1 in a billion) in theory. not so in practice (fails first at cuda30 -n 1679)
#ifndef BIGEPS
#define BIGEPS 3/64
#endif

template<u32 BUCKETSIZE, u32 NRENAME, u32 NRENAME1>
struct zbucket {
  u32 size;
  const static u32 RENAMESIZE = 2*NRENAME1 + 2*NRENAME;
  union {
    u8 bytes[BUCKETSIZE];
    struct {
      u32 words[BUCKETSIZE/sizeof(u32) - RENAMESIZE];
      u32 renameu1[NRENAME1];
      u32 renamev1[NRENAME1];
      u32 renameu[NRENAME];
      u32 renamev[NRENAME];
    };
  };
  __device__ void setsize(u8 const *end) {
    size = end - bytes;
    assert(size <= BUCKETSIZE);
  }
};

const static u32 ZBUCKETSLOTS = NZ + NZ * BIGEPS;
const static u32 ZBUCKETSIZE = ZBUCKETSLOTS * BIGSIZE;

typedef zbucket<ZBUCKETSIZE, NZ1, NZ2> bigbuck;
typedef bigbuck bigbucks[NY];

typedef zbucket<Z2BUCKETSIZE, 0, 0> smallbuck;

struct indexer {
  bigbucks *buckets;
  u32 index[NX];

  __device__ void init(bigbucks *bkts) {
    if (!threadIdx.x)
      buckets = bkts;
  }
  __device__ void matrixu(const u32 x) {
    for (u32 y = threadIdx.x; y < NY; y += blockDim.x)
      index[y] = buckets[x][y].bytes - (u8 *)buckets;
  }
  __device__ void matrixv(const u32 y) {
    for (u32 x = threadIdx.x; x < NX; x += blockDim.x)
      index[x] = buckets[x][y].bytes - (u8 *)buckets;
  }
  template <u32 SIZE>
  __device__ void writebytes(u32 i, const u64 x) {
    memcpy((u8 *)buckets + atomicAdd(index+i, SIZE), (u8 *)&x, SIZE);
  }
  __device__ void write32(u32 i, const u32 x) {
    *(u32 *)((u8 *)buckets + atomicAdd(index+i, sizeof(u32))) = x;
  }
  __device__ void storeu(const u32 x) {
    for (u32 y = threadIdx.x; y < NY; y += blockDim.x)
      buckets[x][y].setsize((u8 *)buckets + index[y]);
  }
  __device__ void storev(const u32 y) {
    for (u32 x = threadIdx.x; x < NX; x += blockDim.x)
      buckets[x][y].setsize((u8 *)buckets + index[x]);
  }
};

template <u32 SIZE>
struct twice_set {
  const static u32 TWICE_WORDS = ((2 * SIZE) / 32);
  u32 bits[TWICE_WORDS];
  __device__ void reset() {
    for (u32 b = threadIdx.x; b < TWICE_WORDS; b += blockDim.x)
      bits[b] = 0;
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit)
      atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define likely(x)   ((x)!=0)
#define unlikely(x) (x)

class edgetrimmer; // avoid circular references

typedef u8 zbucket8[NYZ1*2];
typedef u32 zbucket32[MAXNZNYZ1];

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

struct twostagetpb {
  u16 stage1tpb;
  u16 stage2tpb;
};

struct trimparams {
  u16 nblocks;
  u16 ntrims;
  u16 genUblocks;
  u16 genUtpb;
  twostagetpb genV;
  twostagetpb trim;
  twostagetpb rename[2];
  u16 trim3tpb;
  u16 rename3tpb;
  u16 reportcount;
  u16 reportrounds;
  
  trimparams() {
    ntrims              = 256;
    nblocks             = 128;
    genUblocks          = 128;
    genUtpb             =   8;
    genV.stage1tpb      =  32;
    genV.stage2tpb      = 128;
    trim.stage1tpb      =  32;
    trim.stage2tpb      = 128;
    rename[0].stage1tpb =  32;
    rename[0].stage2tpb =  64;
    rename[1].stage1tpb =  32;
    rename[1].stage2tpb = 128;
    trim3tpb            =  32;
    rename3tpb          =   8;
    reportcount         =   1;
    reportrounds        =   0;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  siphash_keys sip_keys;
  edgetrimmer *dt;
  bigbucks *buckets;
  bigbucks *tbuckets;
  zbucket32 *tnames;
  u32 *uvnodes;
  proof sol;

  edgetrimmer(const trimparams _tp) {
    tp = _tp;
    checkCudaErrors(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors(hipMalloc((void**)&buckets, NX * sizeof(bigbucks)));
    checkCudaErrors(hipMalloc((void**)&tbuckets, tp.nblocks * sizeof(bigbucks)));
    checkCudaErrors(hipMalloc((void**)&tnames, tp.nblocks * sizeof(zbucket32)));
    checkCudaErrors(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
  }
  u64 sharedbytes() const {
    return NX * sizeof(bigbucks);
  }
  u64 threadbytes() const {
    return sizeof(bigbucks) + sizeof(zbucket32);
  }
  ~edgetrimmer() {
    checkCudaErrors(hipFree(buckets));
    checkCudaErrors(hipFree(tbuckets));
    checkCudaErrors(hipFree(tnames));
    checkCudaErrors(hipFree(uvnodes));
  }
  u32 count(u32 size_of) {
    u32 size, sumsize = 0;
    for (u32 ux = 0; ux < tp.reportcount; ux++)
      for (u32 vx = 0; vx < tp.reportcount; vx++) {
        hipMemcpy(&size, &buckets[ux][vx].size, sizeof(u32), hipMemcpyDeviceToHost);
        sumsize += size;
      }
    return sumsize / size_of;
  }

  template <u32 SIZE>
  __device__ void writebytes(u8 *p64, const u64 x) {
    memcpy(p64, (u8 *)&x, SIZE);
  }

  template <u32 SIZE>
  __device__ u64 readbytes(const u8 *p64) {
    u64 foo = 0;
    memcpy((u8 *)&foo, p64, SIZE);
    return foo;
  }

  __device__ void genUnodes(const u32 uorv) {
    __shared__ indexer dst;

    dst.init(buckets);
    for (u32 y = blockIdx.x; y < NY; y += gridDim.x) {
      dst.matrixv(y);
      __syncthreads();
      u32          edge = y << YZBITS;
      const u32 endedge = edge + NYZ;
      for (edge += threadIdx.x; edge < endedge; edge += blockDim.x) {
        const u32 node = dipnode(sip_keys, edge, uorv);
        const u32 ux = node >> YZBITS;
// bit        28..22     21..15    14..0
// node       XXXXXX     YYYYYY    ZZZZZ
        dst.writebytes<BIGSIZE>(ux, (u64)edge << YZBITS | (node & YZMASK));
// bit        39..22     21..15    14..0
// write        edge     YYYYYY    ZZZZZ
      }
      __syncthreads();
      dst.storev(y);
    }
  }

  __device__ void genVnodes1(const u32 part) {
    __shared__ indexer dst;

    dst.init(tbuckets);
    const u32 ux = blockIdx.x + part * gridDim.x;
    dst.matrixu(blockIdx.x);
    __syncthreads();
    for (u32 my = 0 ; my < NY; my++) {
      u32 edge = my << YZBITS;
      bigbuck &zb = buckets[ux][my];
      const u8 *readb = zb.bytes, *endreadb = readb + zb.size;
      for (readb += BIGSIZE*threadIdx.x; readb < endreadb; readb += BIGSIZE*blockDim.x) {
        const u64 e = readbytes<BIGSIZE>(readb);
// bit        39..22     21..15    14..0
// read         edge     UYYYYY    UZZZZ   within UX partition
        const u32 lag = NNONYZ >> 2;
        edge += (((u32)(e >> YZBITS) - edge + lag) & (NNONYZ-1)) - lag;
        const u32 uy = (e >> ZBITS) & YMASK;
        dst.writebytes<BIGSIZE>(uy, ((u64)edge << ZBITS) | (e & ZMASK));;
// bit         39..15     14..0
// write         edge     UZZZZ   within UX UY partition
      }
      if (unlikely(edge >> NONYZBITS != (((my+1) << YZBITS) - 1) >> NONYZBITS))
      { printf("OOPS1: id %d ux %d y %d edge %x vs %x\n", blockIdx.x, ux, my, edge, ((my+1)<<YZBITS)-1); assert(0); }
    }
    __syncthreads();
    dst.storeu(blockIdx.x);
  }

  __device__ void genVnodes2(const u32 part, const u32 uorv) {
    static const u32 NONDEGBITS = (BIGSLOTBITS < 2 * YZBITS ? BIGSLOTBITS : 2 * YZBITS) - ZBITS;
    static const u32 NONDEGMASK = (1 << NONDEGBITS) - 1;
    __shared__ indexer dst;
    __shared__ twice_set<NZ> degs;

    dst.init(buckets);
    const u32 ux = blockIdx.x + part * gridDim.x;
    dst.matrixu(ux);
    for (u32 uy = 0 ; uy < NY; uy++) {
      degs.reset();
      __syncthreads();
      bigbuck &zb = tbuckets[blockIdx.x][uy];
      u8 *readb = zb.bytes, *endreadb = readb + zb.size;
      readb += BIGSIZE * threadIdx.x;
      for (u8 *rd= readb; rd< endreadb; rd+=BIGSIZE*blockDim.x)
        degs.set(readbytes<ZBYTES>(rd) & ZMASK);
      __syncthreads();
      u32 edge = 0;
      u64 uy37 = (u64)uy << YZZBITS;
      for (u8 *rd= readb; rd< endreadb; rd+=BIGSIZE*blockDim.x) {
        const u64 e = readbytes<BIGSIZE>(rd);
// bit         39..15     14..0
// read          edge     UZZZZ    within UX UY partition
        const u32 lag = NONDEGMASK >> 2;
        edge += (((e >> ZBITS) - edge + lag) & NONDEGMASK) - lag;
        const u32 z = e & ZMASK;
        if (degs.test(z)) {
          const u32 node = dipnode(sip_keys, edge, uorv);
          const u32 vx = node >> YZBITS; // & XMASK;
          dst.writebytes<BIGSIZE>(vx, uy37 | ((u64)z << YZBITS) | (node & YZMASK));
// bit        39..37    36..22     21..15     14..0
// write      UYYYYY    UZZZZZ     VYYYYY     VZZZZ   within VX partition
        }
      }
      __syncthreads();
      if (unlikely(edge >> NONDEGBITS != EDGEMASK >> NONDEGBITS))
      { printf("OOPS2: id %d ux %d uy %d edge %x vs %x\n", blockIdx.x, ux, uy, edge, EDGEMASK); assert(0); }
    }
    dst.storeu(ux);
  }

#define mymin(a,b) ((a) < (b) ? (a) : (b))

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
  __device__ void trimedges1(const u32 round, const u32 part) {
    static const u32 SRCSLOTBITS = mymin(SRCSIZE * 8, 2 * YZBITS);
    static const u32 SRCPREFBITS = SRCSLOTBITS - YZBITS;
    static const u32 SRCPREFMASK = (1 << SRCPREFBITS) - 1;
    __shared__ indexer dst;

    dst.init(tbuckets);
    const u32 vx = blockIdx.x + part * gridDim.x;
    dst.matrixu(blockIdx.x);
    for (u32 ux = 0; ux < NX; ux++) {
      __syncthreads();
      u32 uyz = 0;
      bigbuck &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
      const u8 *readbg = zb.bytes, *endreadbg = readbg + zb.size;
      for (readbg += SRCSIZE*threadIdx.x; readbg < endreadbg; readbg += SRCSIZE*blockDim.x) {
        const u64 e = readbytes<SRCSIZE>(readbg); // & SRCSLOTMASK;
// bit     43/39..37    36..22     21..15     14..0
// write      UYYYYY    UZZZZZ     VYYYYY     VZZZZ   within VX partition
        static const u32 lag = SRCPREFMASK >> 2;
        if (SRCPREFBITS >= YZBITS)
          uyz = e >> YZBITS;
        else uyz += (((u32)(e >> YZBITS) - uyz + lag) & SRCPREFMASK) - lag;
        const u32 vy = (e >> ZBITS) & YMASK;
        dst.writebytes<DSTSIZE>(vy, ((u64)(ux << YZBITS | uyz) << ZBITS) | (e & ZMASK));
// bit     43/39..37    36..30     29..15     14..0
// write      UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition
        uyz &= ~ZMASK;
      }
      if (unlikely(uyz >> ZBITS >= NY))
      { printf("OOPS3: id %d vx %d ux %d uyz %x\n", blockIdx.x, vx, ux, uyz); break; }
    }
    __syncthreads();
    dst.storeu(blockIdx.x);
  }

  template <u32 DSTSIZE, bool TRIMONV>
  __device__ void trimedges2(const u32 round, const u32 part) {
    static const u32 DSTSLOTBITS = mymin(DSTSIZE * 8, 2 * YZBITS);
    static const u32 DSTPREFBITS = DSTSLOTBITS - YZZBITS;
    static const u32 DSTPREFMASK = (1 << DSTPREFBITS) - 1;
    __shared__ indexer dst;
    __shared__ twice_set<NZ> degs;

    dst.init(buckets);
    const u32 vx = blockIdx.x + part * gridDim.x;
    TRIMONV ? dst.matrixv(vx) : dst.matrixu(vx);
    for (u32 vy = 0 ; vy < NY; vy++) {
      const u64 vy37 = (u64)vy << YZZBITS;
      degs.reset();
      __syncthreads();
      bigbuck &zb = tbuckets[blockIdx.x][vy];
      u8 *readb = zb.bytes, *endreadb = readb + zb.size;
      readb += DSTSIZE * threadIdx.x;
      for (u8 *rd= readb; rd< endreadb; rd+= DSTSIZE*blockDim.x)
        degs.set(readbytes<ZBYTES>(rd) & ZMASK);
      __syncthreads();
      u32 ux = 0;
      for (u8 *rd= readb; rd< endreadb; rd+= DSTSIZE*blockDim.x) {
        const u64 e = readbytes<DSTSIZE>(rd); //  & DSTSLOTMASK;
// bit     45/39..37    36..30     29..15     14..0
// read       UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition
        if (DSTPREFBITS < XBITS) {
          static const u32 lag = DSTPREFMASK >> 2;
          ux += (((u32)(e >> YZZBITS) - ux + lag) & DSTPREFMASK) - lag;
        } else ux = e >> YZZBITS;
        if (degs.test(e & ZMASK))
          dst.writebytes<DSTSIZE>(ux, vy37 | ((e & ZMASK) << YZBITS) | ((e >> ZBITS) & YZMASK));
// bit    41/39..37    36..22     21..15     14..0
// write     VYYYYY    VZZZZZ     UYYYYY     UZZZZ   within UX partition
      }
      __syncthreads();
      if (unlikely(ux >> DSTPREFBITS != XMASK >> DSTPREFBITS))
      { printf("OOPS4: id %d.%d vx %x ux %x vs %x\n", blockIdx.x, threadIdx.x, vx, ux, XMASK); assert(0); }
    }
    TRIMONV ? dst.storev(vx) : dst.storeu(vx);
  }

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
  __device__ void trimrename1(const u32 round, const u32 part) {
    __shared__ indexer dst;

    dst.init(tbuckets);
    const u32 vx = blockIdx.x + part * gridDim.x;
    dst.matrixu(blockIdx.x);
    for (u32 ux = 0 ; ux < NX; ux++) {
      __syncthreads();
      bigbuck &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
      const u8 *readbg = zb.bytes, *endreadbg = readbg + zb.size;
      for (readbg += SRCSIZE*threadIdx.x; readbg < endreadbg; readbg += SRCSIZE*blockDim.x) {
// bit   43..37    36..22     21..15     14..0
// write UYYYYY    UZZZZZ     VYYYYY     VZZZZ   within VX partition  if TRIMONV
        const u64 e = readbytes<SRCSIZE>(readbg); //  & SRCSLOTMASK;
// bit            37...22     21..15     14..0
// write          VYYYZZ'     UYYYYY     UZZZZ   within UX partition  if !TRIMONV
        const u32 uyz = e >> YZBITS;
        const u32 vy = (e >> ZBITS) & YMASK;
// bit    43..37    36..30     29..15     14..0
// write  UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition  if TRIMONV
        dst.writebytes<SRCSIZE>(vy, ((u64)(ux << (TRIMONV ? YZBITS : YZ1BITS) | uyz) << ZBITS) | (e & ZMASK));
// bit            37...31     30...15     14..0
// write          VXXXXXX     VYYYZZ'     UZZZZ   within UX UY partition  if !TRIMONV
      }
    }
      __syncthreads();
    dst.storeu(blockIdx.x);
  }

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
  __device__ void trimrename2(const u32 round, const u32 part) {
    __shared__ twice_set<NZ> degs;
    __shared__ indexer dst;
    static const u32 NONAME = ~0;

    dst.init(buckets);
    const u32 vx = blockIdx.x + part * gridDim.x;
    TRIMONV ? dst.matrixv(vx) : dst.matrixu(vx);
    u32 *names = tnames[blockIdx.x];
    u32 nrenames = threadIdx.x;
    for (u32 vy = 0 ; vy < NY; vy++) {
      for (u32 z = threadIdx.x; z < NZ; z += blockDim.x)
        names[z] = NONAME;
      degs.reset();
      __syncthreads();
      bigbuck &zb = tbuckets[blockIdx.x][vy];
      u8 *readb = zb.bytes, *endreadb = readb + zb.size;
      readb += SRCSIZE * threadIdx.x;
      for (u8 *rd= readb; rd< endreadb; rd+= SRCSIZE*blockDim.x)
        degs.set(readbytes<ZBYTES>(rd) & ZMASK);
      __syncthreads();
      for (u8 *rd= readb; rd< endreadb; rd+= SRCSIZE*blockDim.x) {
// bit    43..37    36..30     29..15     14..0
// read   UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition  if TRIMONV
        const u64 e = readbytes<SRCSIZE>(rd); //  & SRCSLOTMASK;
// bit            37...31     30...15     14..0
// read           VXXXXXX     VYYYZZ'     UZZZZ   within UX UY partition  if !TRIMONV
        const u32 ux = e >> (TRIMONV ? YZZBITS : YZZ1BITS);
        const u32 vz = e & ZMASK;
        if (degs.test(vz)) {
          u32 vdeg = atomicCAS(&names[vz], NONAME, nrenames);
          if (vdeg == NONAME) {
            vdeg = nrenames;
            if (TRIMONV)
              buckets[vdeg >> Z1BITS][vx].renamev[vdeg & Z1MASK] = vy << ZBITS | vz;
            else
              buckets[vx][vdeg >> Z1BITS].renameu[vdeg & Z1MASK] = vy << ZBITS | vz;
            nrenames += blockDim.x;
          }
          if (TRIMONV)
            dst.writebytes<DSTSIZE>(ux, ((u64)vdeg << YZBITS ) | ((e >> ZBITS) & YZMASK));
// bit       37..22     21..15     14..0
// write     VYYZZ'     UYYYYY     UZZZZ   within UX partition  if TRIMONV
          else dst.write32(ux, (vdeg << YZ1BITS) | ((e >> ZBITS) & YZ1MASK));
        }
      }
      __syncthreads();
    }
    TRIMONV ? dst.storev(vx) : dst.storeu(vx);
    assert(nrenames < NYZ1);
  }

  template <bool TRIMONV>
  __device__ void trimedges3(const u32 round) {
    __shared__ twice_set<NYZ1> degs;

    for (u32 vx = blockIdx.x; vx < NY; vx += gridDim.x) {
      degs.reset();
      __syncthreads();
      for (u32 ux = threadIdx.x ; ux < NX; ux += blockDim.x) {
        bigbuck &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        for (; readbg < endreadbg; readbg++)
          degs.set((*readbg >> (TRIMONV ? 0 : YZ1BITS)) & YZ1MASK);
      }
      __syncthreads();
      for (u32 ux = threadIdx.x ; ux < NX; ux += blockDim.x) {
        bigbuck &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        u32 *write = readbg;
        for (; readbg < endreadbg; readbg++) {
          const u32 e = *readbg;
// bit       31...16     15....0
// read      UYYZZZ'     VYYVZZ'   within VX partition
          const u32 vyz = (e >> (TRIMONV ? 0 : YZ1BITS)) & YZ1MASK;
          if (degs.test(vyz))
            *write++ = e;
        }
        zb.setsize((u8 *)write);
      }
      __syncthreads();
    }
  }

  template <bool TRIMONV>
  __device__ void trimrename3(const u32 round) {
    __shared__ twice_set<NYZ1> degs;
    __shared__ u32 dstidx;
    const u32 NONAME = ~0;

    smallbuck *bucks = (smallbuck *)tbuckets;
    u32 *names = tnames[blockIdx.x];
    for (u32 vx = blockIdx.x; vx < NY; vx += gridDim.x) {
      u32 vx25 = vx << (YZ2BITS + XYZ2BITS);
      __syncthreads();
      for (u32 z = threadIdx.x; z < NYZ1; z += blockDim.x)
        names[z] = NONAME;
      degs.reset();
      __syncthreads();
      for (u32 ux = threadIdx.x; ux < NX; ux += blockDim.x) {
        bigbuck &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        for (; readbg < endreadbg; readbg ++)
          degs.set(*readbg & YZ1MASK);
      }
      u32 nrenames = threadIdx.x;
      smallbuck &vb = bucks[vx];
      if (!TRIMONV && !threadIdx.x)
        dstidx = 0;
      __syncthreads();
      for (u32 ux = threadIdx.x; ux < NX; ux += blockDim.x) {
        bigbuck &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        u32 *write = readbg;
        for (; readbg < endreadbg; readbg ++) {
          const u32 e = *readbg;
// bit      31...16     15...0
// read     UYYYZZ'     VYYZZ'   within VX partition
          const u32 vyz = e & YZ1MASK;
          if (degs.test(vyz)) {
            u32 vdeg = atomicCAS(&names[vyz], NONAME, nrenames);
            if (vdeg == NONAME) {
              vdeg = nrenames;
              if (TRIMONV)
                buckets[vdeg >> Z2BITS][vx].renamev1[vdeg & Z2MASK] = vyz;
              else
                buckets[vx][vdeg >> Z2BITS].renameu1[vdeg & Z2MASK] = vyz;
              nrenames += blockDim.x;
            }
            if (TRIMONV) {
              *write++ = (vdeg  << YZ1BITS) | (e >> YZ1BITS);
// bit      24...16     15...0
// write    VYYZZZ"     UYYZZ'   within UX partition
	    } else {
              vb.words[atomicAdd(&dstidx, 1)] = vx25 | (((vdeg << XBITS) | ux) << YZ2BITS) | (e >> YZ1BITS);
// bit      31...16   15....0
// write    UXXYYZZ"  VXXYYZZ"
            }
          }
        }
        if (TRIMONV) zb.setsize((u8 *)write);
      }
      assert(nrenames < NYZ2);
      __syncthreads();
      if (!TRIMONV && !threadIdx.x) vb.setsize((u8 *)(vb.words+dstidx));
    }
  }

  __device__ void recoveredges() {
    __shared__ u32 u, ux, uyz, v, vx, vyz;

    if (!threadIdx.x) {
      const u32 u1 = uvnodes[2*blockIdx.x], v1 = uvnodes[2*blockIdx.x+1];
      ux = u1 >> YZ2BITS;
      vx = v1 >> YZ2BITS;
      uyz = buckets[ux][(u1 >> Z2BITS) & YMASK].renameu1[u1 & Z2MASK];
      assert(uyz < NYZ1);
      vyz = buckets[(v1 >> Z2BITS) & YMASK][vx].renamev1[v1 & Z2MASK];
      assert(vyz < NYZ1);
      uyz = buckets[ux][uyz >> Z1BITS].renameu[uyz & Z1MASK];
      vyz = buckets[vyz >> Z1BITS][vx].renamev[vyz & Z1MASK];
      u = (ux << YZBITS) | uyz;
      v = (vx << YZBITS) | vyz;
      uvnodes[2*blockIdx.x] = u;
      uvnodes[2*blockIdx.x+1] = v;
    }
    __syncthreads();
  }

  __device__ void recoveredges1() {
    __shared__ u32 uxymap[NXY/32];

    for (u32 i = threadIdx.x; i < PROOFSIZE; i += blockDim.x) {
      const u32 uxy = uvnodes[2*i] >> ZBITS;
      atomicOr(&uxymap[uxy/32], 1 << uxy%32);
    }
    __syncthreads();
    for (u32 edge = blockIdx.x * blockDim.x + threadIdx.x; edge < NEDGES; edge += gridDim.x * blockDim.x) {
      const u32 u = dipnode(sip_keys, edge, 0);
      const u32 uxy = u  >> ZBITS;
      if ((uxymap[uxy/32] >> uxy%32) & 1) {
        for (u32 j = 0; j < PROOFSIZE; j++) {
           if (uvnodes[2*j] == u && dipnode(sip_keys, edge, 1) == uvnodes[2*j+1]) {
             sol[j] = edge;
           }
        }
      }
    }
  }

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV> void _trimedges(edgetrimmer *et, const u32 round);
  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV> void _trimrename(edgetrimmer *et, const u32 round);

  void trim();
};

__global__ void _genUnodes(edgetrimmer *et, const u32 uorv) {
  et->genUnodes(uorv);
}

__global__ void _genVnodes1(edgetrimmer *et, const u32 part) {
  et->genVnodes1(part);
}

__global__ void _genVnodes2(edgetrimmer *et, const u32 part, const u32 uorv) {
  et->genVnodes2(part, uorv);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
__global__ void _trimedges1(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimedges1<SRCSIZE, DSTSIZE, TRIMONV>(round, part);
}

template <u32 DSTSIZE, bool TRIMONV>
__global__ void _trimedges2(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimedges2<DSTSIZE, TRIMONV>(round, part);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
void edgetrimmer::_trimedges(edgetrimmer *et, const u32 round) {
  for (u32 part=0; part < NX/tp.nblocks; part++) {
    _trimedges1<SRCSIZE, DSTSIZE, TRIMONV><<<tp.nblocks,tp.trim.stage1tpb>>>(dt, round, part);
    _trimedges2<         DSTSIZE, TRIMONV><<<tp.nblocks,tp.trim.stage2tpb>>>(dt, round, part);
  }
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
__global__ void _trimrename1(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimrename1<SRCSIZE, DSTSIZE, TRIMONV>(round, part);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
__global__ void _trimrename2(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimrename2<SRCSIZE, DSTSIZE, TRIMONV>(round, part);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
void edgetrimmer::_trimrename(edgetrimmer *et, const u32 round) {
  for (u32 part=0; part < NX/tp.nblocks; part++) {
    _trimrename1<SRCSIZE, DSTSIZE, TRIMONV><<<tp.nblocks,tp.rename[round&1].stage1tpb>>>(dt, round, part);
    _trimrename2<SRCSIZE, DSTSIZE, TRIMONV><<<tp.nblocks,tp.rename[round&1].stage2tpb>>>(dt, round, part);
  }
}

template <bool TRIMONV>
__global__ void _trimedges3(edgetrimmer *et, const u32 round) {
  et->trimedges3<TRIMONV>(round);
}

template <bool TRIMONV>
__global__ void _trimrename3(edgetrimmer *et, const u32 round) {
  et->trimrename3<TRIMONV>(round);
}

__global__ void _recoveredges(edgetrimmer *et) {
  et->recoveredges();
}

__global__ void _recoveredges1(edgetrimmer *et) {
  et->recoveredges1();
}

#ifndef EXPANDROUND
#define EXPANDROUND 5
#endif

#define BIGGERSIZE BIGSIZE+1

  void edgetrimmer::trim() {
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    hipEvent_t start, stop, startall, stopall;
    checkCudaErrors(hipEventCreate(&startall)); checkCudaErrors(hipEventCreate(&stopall));
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
    float duration;
    hipEventRecord(start, NULL);
    _genUnodes<<<tp.genUblocks,tp.genUtpb>>>(dt, 0);
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    if (0 < tp.reportrounds)
      printf("genUnodes size %u completed in %.0f ms\n", count(BIGSIZE), duration);
    hipEventRecord(start, NULL);
    for (u32 part=0; part < NX/tp.nblocks; part++) {
      _genVnodes1<<<tp.nblocks,tp.genV.stage1tpb>>>(dt, part);
      _genVnodes2<<<tp.nblocks,tp.genV.stage2tpb>>>(dt, part, 1);
    }
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    if (1 < tp.reportrounds)
      printf("genVnodes size %u completed in %.0f ms\n", count(BIGSIZE), duration);
    for (u32 round = 2; round < tp.ntrims-2; round += 2) {
      hipEventRecord(start, NULL);
      u32 size_of = BIGGERSIZE;
      if (round < COMPRESSROUND) {
        if (round < EXPANDROUND) {
          _trimedges<BIGSIZE, BIGSIZE, true>(dt, round);
          size_of = BIGSIZE;
        } else if (round == EXPANDROUND) {
          _trimedges<BIGSIZE, BIGGERSIZE, true>(dt, round);
        } else _trimedges<BIGGERSIZE, BIGGERSIZE, true>(dt, round);
      } else if (round==COMPRESSROUND) {
        _trimrename<BIGGERSIZE, BIGGERSIZE, true>(dt, round);
      } else {
        _trimedges3<true><<<tp.nblocks,tp.trim3tpb>>>(dt, round);
        size_of = sizeof(u32);
      }
      checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
      hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
      if (round < tp.reportrounds)
        printf("round %d size %u completed in %.0f ms\n", round, count(size_of), duration);

      hipEventRecord(start, NULL);
      size_of = BIGGERSIZE;
      if (round < COMPRESSROUND) {
        if (round+1 < EXPANDROUND) {
          _trimedges<BIGSIZE, BIGSIZE, false>(dt, round+1);
          size_of = BIGGERSIZE;
        } else if (round+1 == EXPANDROUND) {
          _trimedges<BIGSIZE, BIGGERSIZE, false>(dt, round+1);
        } else _trimedges<BIGGERSIZE, BIGGERSIZE, false>(dt, round+1);
      } else if (round==COMPRESSROUND) {
        _trimrename<BIGGERSIZE, sizeof(u32), false>(dt, round+1);
        size_of = sizeof(u32);
        hipEventRecord(startall, NULL);
      } else {
        _trimedges3<false><<<tp.nblocks,tp.trim3tpb>>>(dt, round+1);
        size_of = sizeof(u32);
      }
      checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
      hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
      if (round+1 < tp.reportrounds)
        printf("round %d size %u completed in %.0f ms\n", round+1, count(size_of), duration);
    }

    hipEventRecord(stopall, NULL); hipEventSynchronize(stopall); hipEventElapsedTime(&duration, startall, stopall);
    if (tp.reportrounds)
      printf("rounds %d through %d completed in %.0f ms\n", COMPRESSROUND+2, tp.ntrims-3, duration);

    hipEventRecord(start, NULL);
    _trimrename3<true ><<<tp.nblocks,tp.rename3tpb>>>(dt, tp.ntrims-2);
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    if (tp.reportrounds)
      printf("trimrename3 round %d size %u completed in %.0f ms\n", tp.ntrims-2, count(sizeof(u32)), duration);
    hipEventRecord(start, NULL);
    _trimrename3<false><<<tp.nblocks,tp.rename3tpb>>>(dt, tp.ntrims-1);
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    if (tp.reportrounds)
      printf("trimrename3 round %d size %u completed in %.0f ms\n", tp.ntrims-1, count(sizeof(u32)), duration);

  }

#define NODEBITS (EDGEBITS + 1)

// grow with cube root of size, hardly affected by trimming
const static u32 MAXPATHLEN = 8 << ((NODEBITS+2)/3);

int nonce_cmp(const void *a, const void *b) {
  return *(u32 *)a - *(u32 *)b;
}

struct solver_ctx {
  edgetrimmer *trimmer;
  smallbuck *buckets;
  u32 *cuckoo;
  u32 uvnodes[2*PROOFSIZE];
  std::bitset<NXY> uxymap;
  std::vector<u32> sols; // concatenation of all proof's indices
  u32 us[MAXPATHLEN];
  u32 vs[MAXPATHLEN];

  solver_ctx(const trimparams tp) {
    trimmer = new edgetrimmer(tp);
    buckets = new smallbuck[NX];
    cuckoo = new u32[CUCKOO_SIZE];
  }
  void setheadernonce(char* const headernonce, const u32 len, const u32 nonce) {
    ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &trimmer->sip_keys);
    sols.clear();
  }
  ~solver_ctx() {
    delete[] cuckoo;
    delete[] buckets;
    delete trimmer;
  }

  void recordedge(const u32 i, const u32 u2, const u32 v2) {
    uvnodes[2*i]   = u2/2;
    uvnodes[2*i+1] = v2/2;
  }

  void solution(const u32 *us, u32 nu, const u32 *vs, u32 nv) {
    u32 ni = 0;
    recordedge(ni++, *us, *vs);
    while (nu--)
      recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
    while (nv--)
    recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
    assert(ni == PROOFSIZE);
    sols.resize(sols.size() + PROOFSIZE);
    hipMemcpy(trimmer->uvnodes, uvnodes, sizeof(uvnodes), hipMemcpyHostToDevice);
    _recoveredges<<<PROOFSIZE,1>>>(trimmer->dt);
    _recoveredges1<<<4096,128>>>(trimmer->dt);
    hipMemcpy(&sols[sols.size() - PROOFSIZE], trimmer->dt->sol, sizeof(trimmer->sol), hipMemcpyDeviceToHost);
    qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), nonce_cmp);
  }

  static const u32 CUCKOO_NIL = ~0;

  u32 path(u32 u, u32 *us) const {
    u32 nu, u0 = u;
    for (nu = 0; u != CUCKOO_NIL; u = cuckoo[u]) {
      if (nu >= MAXPATHLEN) {
        while (nu-- && us[nu] != u) ;
        if (~nu) {
          printf("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
	  exit(0);
	}
        printf("maximum path length exceeded\n");
	return 0; // happens once in a million runs or so; signal trouble
      }
      us[nu++] = u;
    }
    return nu;
  }

  void addedge(u32 uxyz, u32 vxyz) {
    const u32 u0 = uxyz << 1, v0 = (vxyz << 1) | 1;
    if (u0 != CUCKOO_NIL) {
      u32 nu = path(u0, us), nv = path(v0, vs);
      if (!nu-- || !nv--)
        return; // drop edge causing trouble
      // printf("vx %02x ux %02x e %08x uxyz %06x vxyz %06x u0 %x v0 %x nu %d nv %d\n", vx, ux, e, uxyz, vxyz, u0, v0, nu, nv);
      if (us[nu] == vs[nv]) {
        const u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        const u32 len = nu + nv + 1;
        printf("%4d-cycle found\n", len);
        if (len == PROOFSIZE)
          solution(us, nu, vs, nv);
      } else if (nu < nv) {
        while (nu--)
          cuckoo[us[nu+1]] = us[nu];
        cuckoo[u0] = v0;
      } else {
        while (nv--)
          cuckoo[vs[nv+1]] = vs[nv];
        cuckoo[v0] = u0;
      }
    }
  }

  void findcycles() {
    memset(cuckoo, (int)CUCKOO_NIL, CUCKOO_SIZE * sizeof(u32));
    checkCudaErrors(hipMemcpy(buckets, trimmer->tbuckets, NX * sizeof(smallbuck), hipMemcpyDeviceToHost));
    u32 sumsize = 0;
    for (u32 ux = 0; ux < NX; ux++) {
      smallbuck &zb = buckets[ux];
      const u32 size = zb.size / sizeof(u32);
      u32 *readbg = zb.words, *endreadbg = readbg + size;
      for (; readbg < endreadbg; readbg++) {
        const u32 e = *readbg;
        addedge((XYZ2BITS <= 16 ? 0 : ux << YZ2BITS) | (e >> XYZ2BITS), e & XYZ2MASK);
      }
      sumsize += size;
    }
    printf("findcycles completed on %d edges\n", sumsize);
  }

  int solve() {
    trimmer->trim();
    findcycles();
    return sols.size() / PROOFSIZE;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len, timems;
  struct timeval time0, time1;
  int c;

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "sb:c:d:h:k:m:n:r:U:u:V:v:T:t:X:x:Y:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        printf("SYNOPSIS\n  cuda30 [-b blocks] [-d device] [-h hexheader] [-k rounds [-c count]] [-m trims] [-n nonce] [-r range] [-U blocks] [-u threads] [-V threads] [-v threads] [-T threads] [-t threads] [-X threads] [-x threads] [-Y threads] [-y threads] [-Z threads] [-z threads]\n");
        printf("DEFAULTS\n  cuda30 -b %d -d %d -h \"\" -k %d -c %d -m %d -n %d -r %d -U %d -u %d -V %d -v %d -T %d -t %d -X %d -x %d -Y %d -y %d -Z %d -z %d\n", tp.nblocks, device, tp.reportrounds, tp.reportcount, tp.ntrims, nonce, range, tp.genUblocks, tp.genUtpb, tp.genV.stage1tpb, tp.genV.stage2tpb, tp.trim.stage1tpb, tp.trim.stage2tpb, tp.rename[0].stage1tpb, tp.rename[0].stage2tpb, tp.rename[1].stage1tpb, tp.rename[1].stage2tpb, tp.trim3tpb, tp.rename3tpb);
        exit(0);
      case 'b':
        tp.nblocks = atoi(optarg);
        break;
      case 'd':
        device = atoi(optarg);
        break;
      case 'k':
        tp.reportrounds = atoi(optarg);
        break;
      case 'c':
        tp.reportcount = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        tp.ntrims = atoi(optarg) & -2; // make even as required by solve()
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        tp.genUblocks = atoi(optarg);
        break;
      case 'u':
        tp.genUtpb = atoi(optarg);
        break;
      case 'V':
        tp.genV.stage1tpb = atoi(optarg);
        break;
      case 'v':
        tp.genV.stage2tpb = atoi(optarg);
        break;
      case 'T':
        tp.trim.stage1tpb = atoi(optarg);
        break;
      case 't':
        tp.trim.stage2tpb = atoi(optarg);
        break;
      case 'X':
        tp.rename[0].stage1tpb = atoi(optarg);
        break;
      case 'x':
        tp.rename[0].stage2tpb = atoi(optarg);
        break;
      case 'Y':
        tp.rename[1].stage1tpb = atoi(optarg);
        break;
      case 'y':
        tp.rename[1].stage2tpb = atoi(optarg);
        break;
      case 'Z':
        tp.trim3tpb = atoi(optarg);
        break;
      case 'z':
        tp.rename3tpb = atoi(optarg);
        break;
    }
  }

  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  assert(tp.genUtpb <= prop.maxThreadsPerBlock);
  assert(tp.genV.stage1tpb <= prop.maxThreadsPerBlock);
  assert(tp.genV.stage2tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.stage1tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.stage2tpb <= prop.maxThreadsPerBlock);
  assert(tp.rename[0].stage1tpb <= prop.maxThreadsPerBlock);
  assert(tp.rename[0].stage2tpb <= prop.maxThreadsPerBlock);
  assert(tp.rename[1].stage1tpb <= prop.maxThreadsPerBlock);
  assert(tp.rename[1].stage2tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim3tpb <= prop.maxThreadsPerBlock);
  assert(tp.rename3tpb <= prop.maxThreadsPerBlock);
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 10240; dbytes>>=10,dunit++) ;
  printf("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);
  hipSetDevice(device);

  printf("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
  if (range > 1)
    printf("-%d", nonce+range-1);
  printf(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, tp.nblocks); 

  solver_ctx ctx(tp);

  u64 sbytes = ctx.trimmer->sharedbytes();
  u64 tbytes = ctx.trimmer->threadbytes();
  u64 bytes = sbytes + tp.nblocks * tbytes;
  int sunit,tunit,unit;
  for (sunit=0; sbytes >= 10240; sbytes>>=10,sunit++) ;
  for (tunit=0; tbytes >= 10240; tbytes>>=10,tunit++) ;
  for ( unit=0;  bytes >= 10240;  bytes>>=10, unit++) ;
  printf("Using %d%cB bucket memory and %d%cB memory per thread block (%d%cB total)\n",
    sbytes, " KMGT"[sunit], tbytes, " KMGT"[tunit], bytes, " KMGT"[unit], NX);

  u32 sumnsols = 0;
  for (int r = 0; r < range; r++) {
    gettimeofday(&time0, 0);
    ctx.setheadernonce(header, sizeof(header), nonce + r);
    printf("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r,
       ctx.trimmer->sip_keys.k0, ctx.trimmer->sip_keys.k1, ctx.trimmer->sip_keys.k2, ctx.trimmer->sip_keys.k3);
    u32 nsols = ctx.solve();
    gettimeofday(&time1, 0);
    timems = (time1.tv_sec-time0.tv_sec)*1000 + (time1.tv_usec-time0.tv_usec)/1000;
    printf("Time: %d ms\n", timems);

    for (unsigned s = 0; s < nsols; s++) {
      printf("Solution");
      u32* prf = &ctx.sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        printf(" %jx", (uintmax_t)prf[i]);
      printf("\n");
      int pow_rc = verify(prf, &ctx.trimmer->sip_keys);
      if (pow_rc == POW_OK) {
        printf("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          printf("%02x", cyclehash[i]);
        printf("\n");
      } else {
        printf("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
  }
  printf("%d total solutions\n", sumnsols);

  return 0;
}
